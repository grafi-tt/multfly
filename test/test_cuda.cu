#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <string>
#include <vector>

#include <stdio.h>

#include <multfly_ref.h>
#include <multfly_device.cuh>

#define CUDA_CHECK(code) do { \
	if (code != hipSuccess) { \
		throw std::runtime_error(__FILE__ ":" + std::to_string(__LINE__) + " Cuda error" + std::to_string(code)); \
	} \
} while (0)

#define TEST_KEY_NAME "multfly_test"

__global__ void init_key(multfly_key *key, uint64_t global_seed, uint64_t global_ctr) {
	multfly_device_init_by_literal(key, TEST_KEY_NAME, global_seed, global_ctr);
}

__global__ void generate_u32(const multfly_key *key, uint32_t *result0, uint32_t *result1) {
	uint64_t id = blockIdx.x * blockDim.x + threadIdx.x;
	result0[id] = multfly_device_gen32(key, id >> 2, 0);
	result1[id] = multfly_device_gen32(key, id >> 2, 1);
}

int main() {
	int ng = 0;

	CUDA_CHECK(hipSetDevice(0));

	int global_seed = 0;
	int global_ctr = 0;

	multfly_key *d_key;
	CUDA_CHECK(hipMalloc(&d_key, sizeof(multfly_key)));
	init_key<<<1, 4>>>(d_key, global_seed, global_ctr);
	CUDA_CHECK(hipDeviceSynchronize());

	multfly_key h_key, h_key_test;
	CUDA_CHECK(hipMemcpy(&h_key, d_key, sizeof(multfly_key), hipMemcpyDeviceToHost));
	h_key_test = multfly_init_by_literal(TEST_KEY_NAME, global_seed, global_ctr);
	for (int i = 0; i < 8; i++) {
		if (h_key.v_[i] != h_key_test.v_[i]) {
			ng = 1;
			std::cout << "init fail" << std::endl;
			break;
		}
	}

	int len = 8192;
	int n_threads = 128;
	int n_blocks = len / n_threads;

	uint32_t *d_result0;
	uint32_t *d_result1;
	CUDA_CHECK(hipMalloc(&d_result0, sizeof(uint32_t) * len));
	CUDA_CHECK(hipMalloc(&d_result1, sizeof(uint32_t) * len));
	generate_u32<<<n_blocks, n_threads>>>(d_key, d_result0, d_result1);
	CUDA_CHECK(hipDeviceSynchronize());

	std::vector<uint32_t> h_result0(len);
	std::vector<uint32_t> h_result1(len);
	uint32_t h_result_test0[4];
	uint32_t h_result_test1[4];
	CUDA_CHECK(hipMemcpy(h_result0.data(), d_result0, sizeof(uint32_t) * len, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(h_result1.data(), d_result1, sizeof(uint32_t) * len, hipMemcpyDeviceToHost));
	for (int i = 0; i < len; i += 4) {
		multfly_gen32(&h_key_test, i >> 2, 0, h_result_test0);
		multfly_gen32(&h_key_test, i >> 2, 1, h_result_test1);
		for (int j = 0; j < 4; j++) {
			if (h_result0[i + j] != h_result_test0[j]) {
				ng = 1;
				std::cout << "uint32 result0 fail" << std::endl;
				i = len;
				break;
			}
			if (h_result1[i + j] != h_result_test1[j]) {
				ng = 1;
				std::cout << "uint32 result1 fail" << std::endl;
				i = len;
				break;
			}
		}
	}

	if (!ng) {
		std::cout << "ok" << std::endl;
	}
	return ng;
}
